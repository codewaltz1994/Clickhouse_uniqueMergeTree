#include <iostream>
#include <Stopwatch.h>
#include <define.h>
//#include <CpuLZ4Compressor.h>
//#include <GpuLZ4Compressor.h>
#include "UserDefinedFunctions.h"

std::vector<std::complex<float>> Functions::fft(const float * data, size_t size)
{
    hipfftComplex * complex_data = new hipfftComplex[size];
    for (size_t i = 0; i < size; ++i)
    {
        complex_data[i].x = data[i];
        complex_data[i].y = 0;
    }

    hipfftComplex * dfft_data;

    hipMalloc((void **)&dfft_data, size * sizeof(hipfftComplex));

    if (hipGetLastError() != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Failed to allocate\n");
        exit(-1);
    }

    Stopwatch sw;
    hipMemcpy(dfft_data, complex_data, size * sizeof(hipfftComplex), hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Failed to Memcpy\n");
        exit(-1);
    }
    auto ms = sw.elapsedMilliseconds();
    std::cout << "Memcpy time: " << ms << " ms, data size = " << size * sizeof(hipfftComplex) << std::endl;

    hipfftHandle plan;
    if (hipfftPlan1d(&plan, size, HIPFFT_C2C, 1) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "Cuda error: Failed to Handle\n");
        exit(-1);
    }

    sw.restart();
    if (hipfftExecC2C(plan, static_cast<hipfftComplex *>(dfft_data), static_cast<hipfftComplex *>(dfft_data), HIPFFT_FORWARD) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "Cuda error: Failed to exec\n");
        exit(-1);
    }
    ms = sw.elapsedMilliseconds();
    std::cout << "FFT execute time: " << ms << " ms" << std::endl;

    hipDeviceSynchronize();

    sw.restart();
    hipMemcpy(complex_data, dfft_data, size * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to Memcpy\n");
		exit(-1);
	}
    ms = sw.elapsedMilliseconds();
    std::cout << "Memcpy time: " << ms << " ms, data size = " << size * sizeof(hipfftComplex) << std::endl << std::endl;

    std::vector<std::complex<float>> res;
    res.resize(size);

    for (size_t i = 0; i < size; ++i)
    {
        res[i] = {complex_data[i].x, complex_data[i].y};
    }
	delete []complex_data;
	hipFree(dfft_data);
	return res;
}

#if 0
std::vector<std::complex<float>> Functions::fftWithCompress(const std::vector<float> & data)
{
    size_t size = data.size();
    hipfftComplex * complex_data = new hipfftComplex[size];
    for (size_t i = 0; i < size; ++i)
    {
        complex_data[i].x = data[i];
        complex_data[i].y = 0;
    }

    BatchDataCPU batch_data_cpu((char *)complex_data, size * sizeof(hipfftComplex), chunk_size);

    /// compress
    auto compress_data_cpu = CpuLZ4::compress(batch_data_cpu);
    compress_data_cpu.compact();

    Stopwatch sw;

    BatchData compress_data(compress_data_cpu, true, true);
    auto ms = sw.elapsedMilliseconds();
    std::cout << "Memcpy time(with compress): " << ms << " ms, data size = " << compress_data_cpu.data_size()
              << ", compress ratio = " << batch_data_cpu.data_size() * 1.0 / compress_data_cpu.data_size() << std::endl;

    auto decompress_data = GpuLZ4::decompress(compress_data, batch_data_cpu);

    hipfftHandle plan;
    if (hipfftPlan1d(&plan, size, HIPFFT_C2C, 1) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "Cuda error: Failed to Handle\n");
        exit(-1);
    }

    sw.restart();
    if (hipfftExecC2C(
            plan,
            reinterpret_cast<hipfftComplex *>(decompress_data.data()),
            reinterpret_cast<hipfftComplex *>(decompress_data.data()),
            HIPFFT_FORWARD)
        != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "Cuda error: Failed to exec\n");
        exit(-1);
    }
    ms = sw.elapsedMilliseconds();
    std::cout << "FFT execute time(with compress): " << ms << " ms" << std::endl;

    hipDeviceSynchronize();

#    if 0
	sw.restart();
    hipMemcpy(complex_data, decompress_data.data(), size * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to Memcpy\n");
		exit(-1);
	}
	ms = sw.elapsedMilliseconds();
    std::cout << "Memcpy time(with non-compress): " << ms << " ms, data size = " << size * sizeof(hipfftComplex) << std::endl << std::endl;
#    endif
	auto compress_data_new = GpuLZ4::compress(decompress_data);
	
	sw.restart();
	auto compress_data_cpu_new = CpuLZ4::GetBatchDataCPU(compress_data_new, true);
	ms = sw.elapsedMilliseconds();
    std::cout << "Memcpy time(with compress): " << ms << " ms, data size = " << compress_data_cpu_new.data_size() << std::endl;

    auto decompress_data_cpu = CpuLZ4::decompress(compress_data_cpu_new, decompress_data);
	
	auto * complex_data_new = reinterpret_cast<hipfftComplex *>(decompress_data_cpu.data());

    std::vector<std::complex<float>> res;
    res.resize(size);

    for (size_t i = 0; i < size; ++i)
    {
		res[i] = {complex_data_new[i].x, complex_data_new[i].y};
        //res[i] = {complex_data[i].x, complex_data[i].y};
    }

    return res;
}
#endif
