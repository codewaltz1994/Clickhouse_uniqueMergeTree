#include <vector>
#include <Stopwatch.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <util.h>

using T = int;
const size_t vec_size = 1 << 18;
const size_t data_size = vec_size * sizeof(T);

int main()
{
    std::vector<int> vec(vec_size);

    char * device_ptr;
    CUDA_CHECK(hipMalloc(&device_ptr, data_size));

    Stopwatch sw;
    CUDA_CHECK(hipMemcpy(device_ptr, vec.data(), data_size, hipMemcpyHostToDevice));
    auto ms = sw.elapsedNanoseconds();
    std::cout << "Elapsed time of copy data from CPU to GPU: " << ms << " ns" << std::endl;

    sw.restart();
    CUDA_CHECK(hipMemcpy(vec.data(), device_ptr, data_size, hipMemcpyDeviceToHost));
    ms = sw.elapsedNanoseconds();
    std::cout << "Elapsed time of copy data from GPU to CPU: " << ms << " ns" << std::endl;
    CUDA_CHECK(hipFree(device_ptr));
}
