#include <fstream>
#include <iostream>
#include <vector>
#include <BatchData.h>
#include <BatchDataCPU.h>
#include <Stopwatch.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <define.h>
//#include <CpuLZ4Compressor.h>
//#include <GpuLZ4Compressor.h>

constexpr size_t uncompressed_size = 1024 * 1024 * 1024;

void f1()
{
    std::ifstream is("/data/fenglv/final/gpu-executor/ol_cdump.json");
    std::vector<uint8_t> data(uncompressed_size);
    is.read((char *)data.data(), uncompressed_size);

    char * device_buf;
    Stopwatch sw;
    hipMalloc(&(device_buf), uncompressed_size);
    hipMemcpy(device_buf, data.data(), uncompressed_size, hipMemcpyHostToDevice);
    auto ms = sw.elapsedMilliseconds();
    std::cout << "hipMemcpy host to device time = " << ms << " ms" << std::endl;

    sw.restart();
    hipMemcpy(data.data(), device_buf, uncompressed_size, hipMemcpyDeviceToHost);
    ms = sw.elapsedMilliseconds();
    std::cout << "hipMemcpy device to host time = " << ms << " ms" << std::endl;
    hipFree(device_buf);
}

void f2()
{
}
int main()
{
    f1();
    f1();
    f2();
    f2();
    f2();
}
