#include <fstream>
#include <vector>
#include <GpuLZ4Compressor.h>

const size_t uncompressed_data_size = 2048 * 4;
const size_t tuple_size = 2048;

int main()
{
    std::vector<int> vec(tuple_size);
    for (size_t i = 0; i < tuple_size; ++i)
    {
        vec[i] = i;
    }
    uint8_t * src, * dst;
    CUDA_CHECK(hipMalloc(&src, uncompressed_data_size));
    CUDA_CHECK(hipMemcpy(src, vec.data(), uncompressed_data_size, hipMemcpyHostToDevice));

    auto compressed_size = GpuLZ4::compress(src, uncompressed_data_size, dst);

    std::cout << "Finish compress, compressed_size = " << compressed_size << std::endl;

    uint8_t * buf = new uint8_t[8320];
    CUDA_CHECK(hipMemcpy(buf, dst, 8320, hipMemcpyDeviceToHost));

    std::ofstream file("./gpu_compress.txt");
    for (size_t i = 96; i < 8320; ++i)
    {
        int a = *(uint8_t *)(buf + i);
        file << a << " ";
    }

    uint8_t * reverse_dst;
    GpuLZ4::decompress(dst, compressed_size, uncompressed_data_size, reverse_dst);

    std::vector<int> res(tuple_size, 0);
    CUDA_CHECK(hipMemcpy(res.data(), reverse_dst, uncompressed_data_size, hipMemcpyDeviceToHost));
    std::cout << "Finish decompress" << std::endl; //" compressed_size = " << compressed_size << std::endl;

    for (int i = 0; i < tuple_size; ++i)
    {
        if (res[i] != i)
        {
            std::cout << "Test Failed.";
            hipFree(src);
            hipFree(dst);
            hipFree(reverse_dst);
            return 1;
        }
    }
    std::cout << "Test Pass!!!" << std::endl;
    CUDA_CHECK(hipFree(src));
    CUDA_CHECK(hipFree(dst));
    CUDA_CHECK(hipFree(reverse_dst));
    return 0;
}
